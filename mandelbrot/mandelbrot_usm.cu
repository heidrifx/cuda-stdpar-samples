#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include ""
#include <iostream>
#include "../helper.h"
#include <chrono>

#define MAX_ITER 10000

/**
 * @brief CUDA kernel device code to generate a picture of the mandelbrot set
 * 
 * @param pos image array
 * @param width image width
 * @param height  image height
 */
__global__ void calc(int *pos, const uint64_t width, const uint64_t height) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;  // WIDTH
    int col = blockIdx.x * blockDim.x + threadIdx.x;  // HEIGHT
    uint64_t idx = row * width + col,
        n = width * height;

    // return if current index, col or row is out of bounds
    if(col >= width || row >= height || idx >= n) return;

    // c = x0 + iy0
    float x0 = ((float)col / width) * 3.5f - 2.5f;
    float y0 = ((float)row / height) * 3.5f - 1.75f;

    // z = x + iy
    float x = 0.0f;
    float y = 0.0f;
    int iter = 0;
    float xtemp;

    while((x * x + y * y <= 4.0f) && (iter < MAX_ITER)) { 
        // z^2 = x^2 + i2xy - y^2
        // Re(z^2 + c) = x^2 - y^2 +x0
        xtemp = x * x - y * y + x0;

        // Im(z^2 + c) = 2xy + y0
        y = 2.0f * x * y + y0;

        x = xtemp;
        iter++;
    }

    int color = iter * 5;
    if (color >= 256) color = 0;
    pos[idx] = color;
}

/**
 * Host routine
 */
int main(int argc, char *argv[]) {
    if (argc < 2) exit(EXIT_FAILURE);

    // memory in GB
    int memory = std::stoi(argv[1]);
    int factor = floor(std::sqrt(pow(10,9)*memory/sizeof(int))/5000);
    uint64_t height = 5000 * factor,
        width = height,
        n = width * height;

    int* image_buffer;
    printf("Calculating Mandelbrot-Set picture of size %llu x %llu\n", width, height);

    // use usm
    checkCudaErrors(hipMallocManaged(&image_buffer, sizeof(int) * n));

    // start timer
    auto start = std::chrono::steady_clock::now();

    dim3 block_size(16, 16);
    dim3 grid_size(width / block_size.x, height / block_size.y);
    calc<<<grid_size, block_size>>>(image_buffer, width, height);
    hipDeviceSynchronize();

    // stop timer
    auto end = std::chrono::steady_clock::now();
    printf("Total time elapsed: %lims\n", std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count());

    // save image
    if (argc > 2 && std::stoi(argv[2]) == 1) {
        FILE* pgmimg;
        pgmimg = fopen("mandelbrot_cuda.pgm", "wb");
        fprintf(pgmimg, "P2\n");
        fprintf(pgmimg, "%d %d\n", width, height);
        fprintf(pgmimg, "255\n");
        for (int j = 0; j < height; ++j) {
            for (auto i = 0; i < width; ++i)
                fprintf(pgmimg, "%d ", image_buffer[j * height + i]);
            fprintf(pgmimg, "\n");
        }
        fclose(pgmimg);
    }

    checkCudaErrors(hipFree(image_buffer));
}
