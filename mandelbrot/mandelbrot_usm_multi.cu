#include "hip/hip_runtime.h"
/**
 * Compute the mandelbrot set
 */

#include <hip/hip_runtime.h>
#include ""
#include <iostream>
#include "../helper.h"
#include <chrono>

#define MAX_ITER 10000

/**
 * CUDA Kernel Device code
 */
__global__ void calc(int *pos, const ull_int width, const ull_int height) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;  // WIDTH
    int col = blockIdx.x * blockDim.x + threadIdx.x;  // HEIGHT
    ull_int idx = row * width + col,
        n = width * height;

    if(col >= width || row >= height || idx >= n) return;

    float x0 = ((float)col / width) * 3.5f - 2.5f;
    float y0 = ((float)row / height) * 3.5f - 1.75f;

    float x = 0.0f;
    float y = 0.0f;
    int iter = 0;
    float xtemp;
    while((x * x + y * y <= 4.0f) && (iter < MAX_ITER)) { 
        xtemp = x * x - y * y + x0;
        y = 2.0f * x * y + y0;
        x = xtemp;
        iter++;
    }

    int color = iter * 5;
    if (color >= 256) color = 0;
    pos[idx] = color;
}

void mandelbrot(const int factor) {
    ull_int height = 5000 * factor,
        width = height,
        n = width * height;

    int* image_buffer;
    printf("Calculating Mandelbrot-Set picture of size %llu x %llu\n", width, height);

    // start timer
    auto start = std::chrono::steady_clock::now();

    checkCudaErrors(hipMallocManaged(&image_buffer, sizeof(int) * n));

    dim3 block_size(16, 16);
    dim3 grid_size(width / block_size.x, height / block_size.y);
    calc<<<grid_size, block_size>>>(image_buffer, width, height);
    hipDeviceSynchronize();

    // stop timer
    auto end = std::chrono::steady_clock::now();
    printf("Total time elapsed: %lims\n", std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count());

    checkCudaErrors(hipFree(image_buffer));
}

/**
 * Host routine
 */
int main() {
    for(int i = 1; i <= 9; ++i)
        mandelbrot(i);
}
