#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <hip/hip_runtime.h>
#include "../helper.h"
#include <cstdio>
#include <cassert>
#include <ctime>
#include <algorithm>
#include <chrono>

#define TYPE int
#define RANDOM 1
#define N (1 << getSpace(argv[1], sizeof(TYPE), 2))
#define BLOCK_SIZE (1 << 9)

/**
 * @brief CUDA kernel device code for a simple vector reduction
 * 
 * @tparam T data type
 * @param g_idata input array
 * @param g_odata output array
 * @param n input length
 */
template<class T>
__global__ void reduce2(const T *g_idata, T *g_odata, const uint64_t n) {
    // shared memory
    extern __shared__ T sdata0[];
    // threadID & array index
    auto tid = threadIdx.x,
            i = blockIdx.x * blockDim.x + threadIdx.x;

    // check if inside array
    if (i < n) {
        // copy to shared memory
        sdata0[tid] = g_idata[i];
        __syncthreads();

        // reduce
        for (auto s = 1; s < blockDim.x; s *= 2) {
            auto index = 2 * s * tid;
            if (index < blockDim.x) sdata0[index] += sdata0[index + s];
            __syncthreads();
        }

        // copy result from shared to global memory
        if (tid == 0) g_odata[blockIdx.x] = sdata0[0];
    }
}

/**
 * Host routine
 */
int main(int argc, char *argv[]) {
    if (argc < 2) exit(EXIT_FAILURE);
    size_t size_V = N,
            size_R = (size_V + BLOCK_SIZE - 1) / BLOCK_SIZE;
    auto mem_size_V = sizeof(TYPE) * size_V,
            mem_size_R = sizeof(TYPE) * size_R;

    // allocate host memory for vectors;
    auto *h_V = (TYPE *) malloc(mem_size_V),
            *h_R = (TYPE *) malloc(mem_size_R);

    // verify
    assert(h_V && h_R && "Failed to allocate host memory for vectors!\n");

    // init host input vector
    h_V[0] = 0;

#if RANDOM
    srand(time(nullptr));
    for (size_t i = 0; i < size_V; i++)
        h_V[i] = rand() % 200; // random int
#else
    for (size_t i = 1; i < size_V; i++)
        h_V[i] = h_V[i - 1] + 1; // increasing numbers
#endif

    // print some values
    printf("( ");
    for (size_t i = 0; i < std::min(5, N); i++)
        printf("%d ", h_V[i]);
    printf("%s)\n", N > 5 ? "..." : "");

    // allocate device memory
    TYPE *d_V, *d_R;
    checkCudaErrors(hipMalloc((void **) (&d_V), mem_size_V));
    checkCudaErrors(hipMalloc((void **) (&d_R), mem_size_R));

    // start timer
    auto start = std::chrono::steady_clock::now();

    // cpy host -> device
    checkCudaErrors(hipMemcpy(d_V, h_V, mem_size_V, hipMemcpyHostToDevice));

    // launch kernel
    dim3 threads, blocks;

    threads = dim3(1<<9, 1, 1);
    blocks = dim3(size_V / threads.x, 1, 1);
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocks.x, threads.x);
    reduce2<TYPE><<<blocks, threads, threads.x * sizeof(TYPE)>>>(d_V, d_R, N);
    checkCudaErrors(hipGetLastError());

    // cpy device -> host
    checkCudaErrors(hipMemcpy(h_R, d_R, mem_size_R, hipMemcpyDeviceToHost));

    auto end = std::chrono::steady_clock::now();
    printf("Total time elapsed: %lims\n", std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count());

    // print result
    TYPE result = 0.f;
    for (size_t i = 0; i < size_R - 1; i++) {
        result += h_R[i];
    }
    result += h_R[size_R - 1];

    // verify
    TYPE verify = 0, diff = 0;
#if RANDOM
    for (size_t i = 0; i < size_V; i++) verify += h_V[i];
#else
    TYPE n = N - 1;
    verify = (n * n + n) / 2;
#endif
    diff = abs(verify - result);
    printf("Verify: %d, Diff: %d\n", verify, diff);
    printf("%s\n", diff < 1e-4 ? "Result = PASS" : "Result = FAIL");

    // free
    checkCudaErrors(hipFree(d_V));
    checkCudaErrors(hipFree(d_R));
    free(h_V);
    free(h_R);
}
