#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/**
 * Single precision general matrix multiplication
 */

#include <hip/hip_runtime.h>
#include "../helper.h"
#include <cstdio>
#include <cassert>
#include <chrono>

#define TYPE double
#define BLOCK_SIZE (1<<5)

/**
 * CUDA Kernel Device code
 */
template<class T>
__global__ void sgemm(const T *A, const T *B, T *C, const size_t wA, const size_t wB) {
    auto bx = blockIdx.x;
    auto by = blockIdx.y;
    auto tx = threadIdx.x;
    auto ty = threadIdx.y;

    auto aBegin = wA * BLOCK_SIZE * by;
    auto aEnd = aBegin + wA - 1;
    auto aStep = BLOCK_SIZE;

    auto bBegin = BLOCK_SIZE * bx;
    auto bStep = BLOCK_SIZE;

    auto Csub = 0.f;

    for (size_t a = aBegin, b = bBegin; a <= aEnd; a += aStep, b += bStep) {
        __shared__ T Asub[BLOCK_SIZE][BLOCK_SIZE];
        __shared__ T Bsub[BLOCK_SIZE][BLOCK_SIZE];

        Asub[ty][tx] = A[a + wA * ty + tx];
        Bsub[ty][tx] = B[b + wB * ty + tx];

        __syncthreads();

#pragma unroll
        for (size_t k = 0; k < BLOCK_SIZE; ++k) {
            Csub += Asub[ty][k] * Bsub[k][tx];
        }

        __syncthreads();
    }

    auto c = wB * BLOCK_SIZE * by + BLOCK_SIZE * bx;
    C[c + wB * ty + tx] = Csub;
}

/**
 * Host routine
 */
int main() {
    size_t block_size = BLOCK_SIZE;

    dim3 dimsA(16 * 8 * block_size, 16 * 8 * block_size, 1),
            dimsB(16 * 8 * block_size, 16 * 8 * block_size, 1);
    printf("MatrixA(%d,%d), MatrixB(%d,%d)\n", dimsA.x, dimsA.y, dimsB.x, dimsB.y);

    // allocate host memory for matrices A and B
    size_t size_A = dimsA.x * dimsA.y,
            size_B = dimsB.x * dimsB.y;
    size_t mem_size_A = sizeof(TYPE) * size_A,
            mem_size_B = sizeof(TYPE) * size_B;
    auto *h_A = (TYPE *) malloc(mem_size_A),
            *h_B = (TYPE *) malloc(mem_size_B);

    // verify
    assert(h_A && h_B && "Failed to allocate host memory for matrices A or B!\n");

    // init host memory
    for (size_t i = 0; i < size_A; ++i) {
        h_A[i] = 1.f;
    }
    for (size_t i = 0; i < size_B; ++i) {
        h_B[i] = 0.01f;
    }

    // allocate host matrix C
    dim3 dimsC(dimsB.x, dimsA.y, 1);
    size_t mem_size_C = dimsC.x * dimsC.y * sizeof(TYPE);
    auto *h_C = (TYPE *) malloc(mem_size_C);

    // verify
    assert(h_C && "Failed to allocate host matrix C!\n");

    // allocate device memory
    TYPE *d_A, *d_B, *d_C;
    checkCudaErrors(hipMalloc((void **) (&d_A), mem_size_A));
    checkCudaErrors(hipMalloc((void **) (&d_B), mem_size_B));
    checkCudaErrors(hipMalloc((void **) (&d_C), mem_size_C));

    // start timer
    auto start = std::chrono::steady_clock::now();

    // cpy host -> device
    checkCudaErrors(hipMemcpy(d_A, h_A, mem_size_A, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_B, h_B, mem_size_B, hipMemcpyHostToDevice));

    // launch kernel
    dim3 threads(block_size, block_size),
            grid(dimsB.x / threads.x, dimsA.y / threads.y);

    sgemm<TYPE><<<grid, threads>>>(d_A, d_B, d_C, dimsA.x, dimsB.x);
    checkCudaErrors(hipGetLastError());

    // run kernel multiple times
    /*
    auto nIter = 1<<12;
    for (size_t i = 0; i < nIter; i++) {
        sgemm<<<grid, threads>>>(d_A, d_B, d_C, dimsA.x, dimsB.x);
    }
    */

    // cpy device -> host
    checkCudaErrors(hipMemcpy(h_C, d_C, mem_size_C, hipMemcpyDeviceToHost));

    auto end = std::chrono::steady_clock::now();
    printf("Total time elapsed: %lims\n", std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count());

    // verify
    bool correct = true;
    double eps = 1.e-6;
    for (int i = 0; i < (int) (dimsC.x * dimsC.y); i++) {
        auto abs_err = fabs(h_C[i] - (dimsA.x * 0.01f));
        auto dot_length = dimsA.x;
        auto abs_val = fabs(h_C[i]);
        auto rel_err = abs_err / abs_val / dot_length;

        if (rel_err > eps) {
            printf("Error! Matrix[%05d]=%.8f, ref=%.8f error term is > %E\n",
                   i, h_C[i], dimsA.x * 0.01f, eps);
            correct = false;
        }
    }
    printf("%s\n", correct ? "Result = PASS" : "Result = FAIL");

    // print result matrix
    /*
    for (size_t i = 0; i < dimsC.x * dimsC.y; i += dimsC.x) {
        printf("(");
        for (size_t j = 0; j < dimsC.x - 1; j++) {
            printf("%f ", h_C[i + j]);
        }
        printf("%f)\n", h_C[i + dimsC.x]);
    }
    */

    // free
    checkCudaErrors(hipFree(d_A));
    checkCudaErrors(hipFree(d_B));
    checkCudaErrors(hipFree(d_C));
    free(h_A);
    free(h_B);
    free(h_C);
}
